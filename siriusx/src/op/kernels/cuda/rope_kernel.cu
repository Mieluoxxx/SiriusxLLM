#include "hip/hip_runtime.h"
/*
 * @Author: Morgan Woods weiyiding0@gmail.com
 * @Date: 2025-03-06 10:32:12
 * @LastEditors: Morgan Woods weiyiding0@gmail.com
 * @LastEditTime: 2025-03-07 14:40:25
 * @FilePath: /SiriusxLLM/siriusx/src/op/kernels/cuda/rope_kernel.cu
 * @Description:
 */
#include "rope_kernel.cuh"

namespace kernel {

__global__ void sin_cos_calc(int head_size, int max_seq_len, float* sin_cache,
                             float* cos_cache) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int head_dim = idx % head_size;
    for (int pos = 0; pos < max_seq_len; pos++) {
        float freq = 1.0f / pow(10000.0f, static_cast<float>(head_dim) /
                                              static_cast<float>(head_size));
        float val = static_cast<float>(pos) * freq;
        float fcr = cosf(val);
        float fci = sinf(val);
        *(sin_cache + pos * head_size + head_dim) = fci;
        *(cos_cache + pos * head_size + head_dim) = fcr;
    }
}

void sin_cos_cache_calc_cuda(int head_size, int max_seq_len,
                             const tensor::Tensor& sin_cache,
                             const tensor::Tensor& cos_cache,
                             hipStream_t stream) {
    CHECK_EQ(sin_cache.is_empty(), false);
    CHECK_EQ(cos_cache.is_empty(), false);

    int threads = head_size;
    if (stream) {
        sin_cos_calc<<<1, threads, 0, stream>>>(
            head_size, max_seq_len, const_cast<float*>(sin_cache.ptr<float>()),
            const_cast<float*>(cos_cache.ptr<float>()));
    } else {
        sin_cos_calc<<<1, threads>>>(
            head_size, max_seq_len, const_cast<float*>(sin_cache.ptr<float>()),
            const_cast<float*>(cos_cache.ptr<float>()));
    }
}

__device__ void rope_calc(float fcr, float fci, float* vec, int32_t idx) {
    float2* vec_ptr = reinterpret_cast<float2*>(vec + idx);
    float2 vec_value = *vec_ptr;
    *vec_ptr = make_float2(vec_value.x * fcr - vec_value.y * fci,
                           vec_value.x * fci + vec_value.y * fcr);
}

__global__ void rope_kernel_cuda_fp32(int pos, int dim, int kv_dim,
                                      int head_size, const float* input_q,
                                      const float* input_k,
                                      const float* sin_cache,
                                      const float* cos_cache) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    idx = idx * 2;
    if (idx >= dim) return;

    int head_dim = idx % head_size;
    float fci = *(sin_cache + pos * head_size + head_dim);
    float fcr = *(cos_cache + pos * head_size + head_dim);

    rope_calc(fcr, fci, const_cast<float*>(input_q), idx);
    if (idx >= kv_dim) return;
    rope_calc(fcr, fci, const_cast<float*>(input_k), idx);
}

void rope_kernel_cuda(int32_t dim, int32_t kv_dim, int32_t head_size,
                      const tensor::Tensor& input_q,
                      const tensor::Tensor& input_k,
                      const tensor::Tensor& input_pos,
                      const tensor::Tensor& sin_cache,
                      const tensor::Tensor& cos_cache, void* stream) {
    const int32_t pos = *input_pos.ptr<int32_t>(0);
    int threads = 128;
    int blocks = (head_size + threads - 1) / threads;
    if (stream) {
        hipStream_t stream_ = static_cast<hipStream_t>(stream);
        rope_kernel_cuda_fp32<<<blocks, threads, 0, stream_>>>(
            pos, dim, kv_dim, head_size, input_q.ptr<float>(),
            input_k.ptr<float>(), sin_cache.ptr<float>(),
            cos_cache.ptr<float>());
    } else {
        rope_kernel_cuda_fp32<<<blocks, threads>>>(
            pos, dim, kv_dim, head_size, input_q.ptr<float>(),
            input_k.ptr<float>(), sin_cache.ptr<float>(),
            cos_cache.ptr<float>());
    }
}
}  // namespace kernel